#ifndef _PINNED_VECTOR_CU
#define _PINNED_VECTOR_CU

#include "pinnedVector.cuh"
	
template <class T>
pinnedVector<T>::pinnedVector(std::size_t size){
  _size = size; hipHostAlloc( (void**)&pinned_ptr, _size*sizeof(T),hipHostMallocDefault );
}
	
template <class T>
pinnedVector<T>::pinnedVector(std::size_t size, T & val){
  _size = size; 
  hipHostAlloc( (void**)&pinned_ptr, _size*sizeof(T),hipHostMallocDefault );
  for (int i = 0; i < _size; i++){pinned_ptr[i] = val;}
}
	
template <class T>
pinnedVector<T>::~pinnedVector(){
  hipHostFree(pinned_ptr);
}
	
template <class T>	
std::size_t pinnedVector<T>::size(){
  return _size;
}

template <class T>
void pinnedVector<T>::alloc(std::size_t size){
_size = size; hipHostAlloc( (void**)&pinned_ptr, _size*sizeof(T),hipHostMallocDefault );
}

template <class T>
void pinnedVector<T>::alloc(std::size_t size, T & val){
  _size = size; 
  hipHostAlloc( (void**)&pinned_ptr, _size*sizeof(T),hipHostMallocDefault );
  for (int i = 0; i < _size; i++){pinned_ptr[i] = val;}
}
	
template <class T>
T& pinnedVector<T>::operator [] (int i){
  return pinned_ptr[i];
}
		
template <class T>
void pinnedVector<T>::copyToDevice(cudaVector<T> & dev, hipStream_t & stream){
	hipMemcpyAsync( dev.getPointer(), pinned_ptr, _size*sizeof(T), hipMemcpyHostToDevice, stream );
}
	
template <class T>
void pinnedVector<T>::copyFromDevice(cudaVector<T> & dev, hipStream_t & stream){
  hipMemcpyAsync( pinned_ptr, dev.getPointer(), _size*sizeof(T), hipMemcpyDeviceToHost, stream );
}
		
template <class T>	
void pinnedVector<T>::operator= (std::vector<T> & v){
  int iter;
  if (v.size() >= _size){ iter = _size; }
  else {iter = v.size();}
		
  for (int i = 0; i < iter; i++){
    pinned_ptr[i] = v[i];
  }

}
	
template <class T>	
void pinnedVector<T>::operator= (cudaVector<T> & v){
	
  int iter;
  if (v.size() >= _size){ iter = _size; }
  else {iter = v.size();}
	
  for (int i = 0; i < iter; i++){
    pinned_ptr[i] = v[i];
  }
}

template <class T>
void pinnedVector<T>::copyTo(std::vector<T> & v){

  int iter;
  if (v.size() >= _size){ iter = _size; }
  else {iter = v.size();}
	
  for (int i = 0; i < iter; i++){
    v[i] = pinned_ptr[i];
  }
}
	
template <class T>	
T* pinnedVector<T>::getPointer(){
  return pinned_ptr;
}
		
#endif
