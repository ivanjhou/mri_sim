#include "hip/hip_runtime.h"
#include "kernelSetup.cuh"

__global__ void setup_kernel( hiprandState * state, int seed )
{
    int tid = threadIdx.x + blockIdx.x*SIM_THREADS;
    hiprand_init( seed, tid, 0, &state[tid] );
}
