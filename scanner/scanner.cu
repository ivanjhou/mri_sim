#include <chrono>
#include "scanner.cuh"
#include "../primitives/lattice.cuh"

Scanner::Scanner(	
	Sequence& sequence,
	Coil& coil,
	SimuParams& params) 
	: 
	sequence(&sequence),
	coil(&coil),
	params(&params)
	{
	scan_k = new kSpace(params.res_x, params.res_y);
	lattice_present = false;
}

Scanner::Scanner(
	Sequence& sequence,
	Coil& coil,
	SimuParams& params,
	Lattice& lattice)
	:
	sequence(&sequence),
	coil(&coil),
	params(&params),
	lattice(&lattice)
	{
	scan_k = new kSpace(params.res_x, params.res_y);
	lattice_present = true;
}

bool Scanner::scan(){
	bool scan_success;

	if (lattice_present){
		scan_success = scan_lattice();
	} else {
		scan_success = scan_single_basis();
	}

	return scan_success;
}

bool Scanner::scan_lattice(){
	printf("Building basis pointer array.\n");
	lattice->setBasisSize(primitives.size());
	printf("Detected %d basis objects\n", lattice->getBasisSize());
	Primitive** basis_dev_pointers[lattice->getBasisSize()];
	for (int i = 0; i < primitives.size(); i++){
		basis_dev_pointers[i] = primitives[i]->devPointer();
	}

	hipStream_t streams[sequence->getNSubSequences()];
	printf("Starting scan.\n");
	printf("Number of sub sequences = %d.\n", sequence->getNSubSequences());
	//Scan* scans = new Scan[sequence->getNSubSequences()];
	Scan* scans[sequence->getNSubSequences()];

	for (int i = 0; i< sequence->getNSubSequences(); i++){
		acqs.push_back(new magAcquisition(params, sequence->getSubSequences(i)));
		hipStreamCreate(&streams[i]);
		scans[i] = new Scan(acqs[i], params, sequence->getSubSequences(i), lattice, basis_dev_pointers, coil->devPointer(), 0, std::vector<int>(15), streams[i]);
	}

	hipDeviceSynchronize();

	long start = std::chrono::duration_cast< std::chrono::milliseconds >(std::chrono::system_clock::now().time_since_epoch()).count();
	for (int i = 0; i < sequence->getNSubSequences(); i++){
		printf("Running scan %d/%d\n",i,sequence->getNSubSequences()-1);
		scans[i]->run_scan_lattice();
	}

	safe_cuda(hipDeviceSynchronize(), "Post Scan\n");
	long end = std::chrono::duration_cast< std::chrono::milliseconds >(std::chrono::system_clock::now().time_since_epoch()).count();
	printf("Simulation Kernel Time: %ld\n", end-start);
	for (int i = 0; i < sequence->getNSubSequences(); i++){
		scans[i]->saveScan();
		hipDeviceSynchronize();
		make_k_space(acqs[i], sequence->getSubSequences(i));
	}

	hipDeviceSynchronize();

	scan_k->get_fft();

	return true;
}

bool Scanner::scan_single_basis(){
	hipStream_t streams[sequence->getNSubSequences()];
	printf("Starting scan.\n");
	printf("Number of sub sequences = %d.\n", sequence->getNSubSequences());
	//Scan* scans = new Scan[sequence->getNSubSequences()];
	Scan* scans[sequence->getNSubSequences()];

	for (int i = 0; i< sequence->getNSubSequences(); i++){
		acqs.push_back(new magAcquisition(params, sequence->getSubSequences(i)));
		hipStreamCreate(&streams[i]);
		scans[i] = new Scan(acqs[i], params, sequence->getSubSequences(i), primitives[0], coil->devPointer(), 0, std::vector<int>(15), streams[i]);
	}

	hipDeviceSynchronize();
	long start = std::chrono::duration_cast< std::chrono::milliseconds >(std::chrono::system_clock::now().time_since_epoch()).count();
	for (int i = 0; i < sequence->getNSubSequences(); i++){
		printf("Running scan %d/%d\n",i,sequence->getNSubSequences()-1);
		scans[i]->runScan();
	}

	safe_cuda(hipDeviceSynchronize(), "Post Scan\n");
	long end = std::chrono::duration_cast< std::chrono::milliseconds >(std::chrono::system_clock::now().time_since_epoch()).count();
	printf("Simulation Kernel Time: %ld\n", end-start);

	for (int i = 0; i < sequence->getNSubSequences(); i++){
		scans[i]->saveScan();
		hipDeviceSynchronize();
		make_k_space(acqs[i], sequence->getSubSequences(i));
	}

	hipDeviceSynchronize();

	scan_k->get_fft();

	return true;
}

bool Scanner::scanCPU(){
	printf("Starting scan.\n");
	printf("Number of sub sequences = %d.\n", sequence->getNSubSequences());
	//Scan* scans = new Scan[sequence->getNSubSequences()];
	ScanCPU* scans[sequence->getNSubSequences()];

	for (int i = 0; i< sequence->getNSubSequences(); i++){
		acqs.push_back(new magAcquisition(params, sequence->getSubSequences(i)));
		scans[i] = new ScanCPU(acqs[i], params, sequence->getSubSequences(i), primitives[0], coil);
	}

	long start = std::chrono::duration_cast< std::chrono::milliseconds >(std::chrono::system_clock::now().time_since_epoch()).count();
	for (int i = 0; i < sequence->getNSubSequences(); i++){
		printf("Running scan %d/%d\n",i,sequence->getNSubSequences()-1);
		scans[i]->runScan();
	}

	long end = std::chrono::duration_cast< std::chrono::milliseconds >(std::chrono::system_clock::now().time_since_epoch()).count();
	printf("Simulation Kernel Time: %ld\n", end-start);

	for (int i = 0; i < sequence->getNSubSequences(); i++){
		scans[i]->saveScan();
		make_k_space(acqs[i], sequence->getSubSequences(i));
	}

	scan_k->get_fft();

	return true;
}

bool Scanner::add_primitive(Primitive& new_primitive){
	primitives.push_back(&new_primitive);
	
	return true;
}

bool Scanner::make_k_space(magAcquisition *acq, const Sequence *seq){
	int kx;
	int ky;

	for (int i = seq->get_k_start(); i < seq->get_k_end(); i++){
		kx = seq->getK(i).x;
		ky = seq->getK(i).y;

		scan_k->set_Mx(kx, ky, (1-2*((kx+ky)&1))*acq->get_signal_x()[i%seq->getReadSteps()]);
		scan_k->set_My(kx, ky, (1-2*((kx+ky)&1))*acq->get_signal_y()[i%seq->getReadSteps()]);
	}

	return true;
}
